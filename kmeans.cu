#include "hip/hip_runtime.h"
#include <unistd.h>
#include <algorithm>
#include <cstdlib>
#include <limits>
#include "exclusiveScan.cu_inl"
#include "cycletimer.h"
//#include <random>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"
#define CHANNEL_NUM 3
#define BLOCK_SIDE 1024
#define BLOCK_SIZE 1024
#define SCAN_BLOCK_DIM BLOCK_SIZE 

struct Point;
//using DataFrame = std::vector<Point>;

struct Point {
    double x, y;     // coordinates
    int cluster;     // no default cluster
    double min_dist;  // default infinite dist to nearest cluster
    double r, g, b;

    __device__ Point() : 
        x(0.0), 
        y(0.0),
        r(0.0),
        g(0.0),
        b(0.0),
        cluster(-1),
        min_dist(__DBL_MAX__) {}
        
    __device__ Point(double x, double y, double r, double g, double b) : 
        x(x), 
        y(y),
        r(r),
        g(g),
        b(b),
        cluster(-1),
        min_dist(__DBL_MAX__) {}

    __device__ double euclid_distance(Point p) {
        return (p.x - x) * (p.x - x) + (p.y - y) * (p.y - y);
    }

    __device__ double color_distance(Point p){
        double v = (p.r - r) * (p.r - r) + (p.b - b) * (p.b - b) + (p.g - g) * (p.g - g);
    
        //printf("CD - r: %f, g: %f, b: %f, r2: %f, g2: %f, b2: %f, v: %f\n",
        //r, g, b, p.r, p.b, p.g, v);
        return v;
    }
};
void raw_print(uint8_t *rgb_image, int width, int height){
    int l = width*height*CHANNEL_NUM;
    float x, y;
    int factor = (width*CHANNEL_NUM);
    for(int i = 0; i < l; i+=CHANNEL_NUM){
        y = (float) (i/factor);
        x = (float) (i%factor);
        printf("X: %f, Y: %f, R: %d, G: %d, B: %d\n", 
        x, y, rgb_image[i], rgb_image[i+1], rgb_image[i+2]);
    }
}
/*
Point* get_df(uint8_t *rgb_image, int width, int height){
    
    int l = width*height*CHANNEL_NUM;
    Point* points = (Point*)malloc(sizeof(Point) * width * height);
    double x, y;
    double r, g, b;
    int factor = (width*CHANNEL_NUM);
    for(int i = 0; i < l; i+=CHANNEL_NUM){
        y = (float) (i/factor);
        x = (float) (i%factor);
        r = rgb_image[i]; 
        g = rgb_image[i+1];
        b = rgb_image[i+2];
        points[i] = (Point(x, y, r, g, b));
    }
    return points;
}
*/

__global__ void update_mean(Point* means, Point* data, size_t* assignments, 
    int total_num_points, int k){
    int one_d_id = blockIdx.x * blockDim.x + threadIdx.x;
    if(one_d_id < k){
    int counts = 0;
    Point p;
    for (size_t point = 0; point < total_num_points; ++point) {
        if(assignments[point] == one_d_id){
            p.x += data[point].x;
            p.y += data[point].y;
            p.r += data[point].r;
            p.g += data[point].g;
            p.b += data[point].b;
            counts += 1;
            
        }
    }
    means[one_d_id].x = p.x / counts;
    means[one_d_id].y = p.y / counts;
    means[one_d_id].r = p.r / counts;
    means[one_d_id].g = p.g / counts;
    means[one_d_id].b = p.b / counts;
    //: %d, %f, %f, %f\n", one_d_id, 
    //means[one_d_id].r,means[one_d_id].g,means[one_d_id].b);}

    //printf("P: x - %d, y - %d, r - %d, g - %d, b - %d, count - %d\n", p.x, p.y,
//p.r, p.g, p.b, counts);
    }
}

//assignments, means, height, width, index_x, index_y
__global__ void set_assignments(Point* data, size_t*  assignments, Point* means, int k, int width, int height ){
    int point = blockIdx.x * blockDim.x + threadIdx.x;
    int total_num_points = width * height;
    if(point < total_num_points){
    //if(data[point].b != 0){printf("PCV: %d, %f\n", point, data[point].b);}
    //if(point < k){printf("MV: %d, %f\n", point, means[point].b);}

    int assignment = 0;
    Point p,m;
    double best_distance = CHANNEL_NUM*256*256;
    //std::numeric_limits<double>::max();
    size_t best_cluster = 0;
    for (size_t cluster = 0; cluster < k; ++cluster) {
        p = data[point];
        m = means[cluster];
        //printf("P: x - %d, y - %d, r - %d, g - %d, b - %d\n", p.x, p.y,
//p.r, p.g, p.b);
        //printf("M: x - %d, y - %d, r - %d, g - %d, b - %d\n", m.x, m.y,
//m.r, m.g, m.b);
        double distance = p.color_distance(m);
        //printf("D: %d\n", distance);
        if (distance < best_distance) {
            //if(point==0)printf("BD: %f, D: %f\n", best_distance, distance);
            best_distance = distance;
            best_cluster = cluster;
            assignment = best_cluster;
        }
    }
    //if(point==0)printf("BD: %f\n", best_distance);
    assignments[point] = assignment; 
    }
}

__global__ void fill_points(Point* points, Point* means, size_t* assignments, int number_of_iterations, int k, int height, int width, 
    uint8_t* rgb_image, uint8_t* new_img, int* init_mean_nums){
    int point = blockIdx.x * blockDim.x + threadIdx.x;
    int total_num_points = width * height;
    double x, y;
    double r, g, b;
    int point_channel = point*CHANNEL_NUM;
    if(point<total_num_points){
        int factor = (width*CHANNEL_NUM);
        y = (double)(point_channel/factor);
        x = (double)(point_channel%factor);
        r = rgb_image[point_channel]; 
        g = rgb_image[point_channel+1];
        b = rgb_image[point_channel+2];
        points[point] = (Point(x, y, r, g, b));
    }
    //if(points[t].b != 0){//printf("PCV: %d, %f\n", point, points[t].b);}
    
}

__global__ void set_new_img(Point* points, Point* means, size_t* assignments, int number_of_iterations, int k, int height, int width, 
    uint8_t* rgb_image, uint8_t* new_img, int* init_mean_nums){
    int point = blockIdx.x * blockDim.x + threadIdx.x;
    int total_num_points = width * height;
    Point p;
    int c;
    //TODO check len
    if(point<total_num_points)
    {
        c = assignments[point];
        p = means[c];
        ////printf("%d\n", point);
        new_img[CHANNEL_NUM*point] = p.r;
        new_img[CHANNEL_NUM*point+1] = p.g;
        new_img[CHANNEL_NUM*point+2] = p.b;
    }
}

__global__ void set_means_init(Point* points, Point* means, size_t* assignments, int number_of_iterations, int k, int height, int width, 
    uint8_t* rgb_image, uint8_t* new_img, int* init_mean_nums){
    int point = blockIdx.x * blockDim.x + threadIdx.x;
    int t = (int) (point/CHANNEL_NUM);
    Point m;
    if(point < k){
        int init_ind = point*((height*width)/k);
        means[point] = points[init_ind];
        /*if(point==0) means[point] = Point(0,0,89,116,58);
        if(point==1) means[point] = Point(0,0,215,192,123);
        if(point==2) means[point] = Point(0,0,50,50,29);
        if(point==3) means[point] = Point(0,0,127,80,42);
        */
        m = means[point];
        //printf("M: x - %f, y - %f, r - %f, g - %f, b - %f\n", m.x, m.y,
//m.r, m.g, m.b);
    }
    
}
void k_means_main(dim3 kgridDim, dim3 kthreadsPerBlock, dim3 gridDim, dim3 threadsPerBlock, Point* points, Point* means, size_t* assignments, int number_of_iterations, int k, int height, int width, 
    uint8_t* rgb_image, uint8_t* new_img, int* init_mean_nums){ 
    int total_num_points = width*height;
    fill_points<<<gridDim, threadsPerBlock>>>(points,means,assignments,number_of_iterations,
    k,height,width,rgb_image,new_img,init_mean_nums);
    //hipDeviceSynchronize();
    set_means_init<<<kgridDim, kthreadsPerBlock>>>(points,means,assignments,number_of_iterations,
    k,height,width,rgb_image,new_img,init_mean_nums);
    //hipDeviceSynchronize();
    for(int i = 0; i< number_of_iterations; i++){
        set_assignments<<<gridDim, threadsPerBlock>>>(points, assignments, means, 
        k, width, height);
        //hipDeviceSynchronize();
        update_mean<<<kgridDim, kthreadsPerBlock>>>(means, points, assignments, 
        total_num_points, k);
        //hipDeviceSynchronize();
    }
    set_new_img<<<gridDim, threadsPerBlock>>>(points,means,assignments,number_of_iterations,
    k,height,width,rgb_image,new_img,init_mean_nums);
    //hipDeviceSynchronize();
}

/*
__global__ void k_means_kernel(Point* points, Point* means, size_t* assignments, int number_of_iterations, int k, int height, int width, 
    uint8_t* rgb_image, uint8_t* new_img, int* init_mean_nums){ 
    int point = blockIdx.x * blockDim.x + threadIdx.x;
    ////printf("%d", point);
    int total_num_points = width * height;
    double x, y;
    uint8_t r, g, b;
    int point_channel = point*CHANNEL_NUM;
    if(point<total_num_points){
        int factor = (width*CHANNEL_NUM);
        y = (double)(point_channel/factor);
        x = (double)(point_channel%factor);
        r = rgb_image[point_channel]; 
        g = rgb_image[point_channel+1];
        b = rgb_image[point_channel+2];
        points[point] = (Point(x, y, r, g, b));
    }
    
    hipDeviceSynchronize();
    if(point < k){
        means[point] = Point(0,0,255,255,255);
    }
    hipDeviceSynchronize();
    
    for(int i = 0; i< number_of_iterations; i++){
        if(point < total_num_points){
            set_assignments(points, assignments, means, point, k, width, height);
        }
        hipDeviceSynchronize();
        // now parallelize over clusters
        // TODO USE SCAN 
        int id = point;
        if(id < k){
            update_mean(means, points, assignments, id, total_num_points);
        }
        hipDeviceSynchronize();
    }

    Point p;
    int c;
    //TODO check len
    if(point<total_num_points)
    {
        c = assignments[point];
        p = points[c];
        ////printf("%d\n", point);
        new_img[CHANNEL_NUM*point] = p.r;
        new_img[CHANNEL_NUM*point+1] = p.g;
        new_img[CHANNEL_NUM*point+2] = p.b;

    }
    hipDeviceSynchronize();
}
*/
/*
void print_df(Points* &points, int width, int height){
    int l = width*height;
    Point p;
    //printf("Size: %d\n", points.size());
    for(int i = 0; i < l; i++){
        p = points[i];
        //printf("X: %f, Y: %f, R: %f, G: %f, B: %f\n",
                p.x, p.y, p.r, p.g, p.b);
    }
    //printf("Printed DF\n");
}
*/


void k_means(uint8_t* rgb_image, int width, int height,
                  size_t k,
                  size_t number_of_iterations) {
    dim3 threadsPerBlock(BLOCK_SIDE, 1, 1);
    const int NUM_BLOCKS_X = (width*height+threadsPerBlock.x-1)/threadsPerBlock.x;
    const int NUM_BLOCKS_Y = 1;
    //(height+threadsPerBlock.y-1)/threadsPerBlock.y;
    dim3 gridDim(NUM_BLOCKS_X , NUM_BLOCKS_Y, 1);
    int KBLOCK_SIDE = k;
    dim3 kgridDim(1 , NUM_BLOCKS_Y, 1);
    dim3 kthreadsPerBlock(KBLOCK_SIDE, 1, 1);
    
    //TODO CUDA RANDOM MEANS and assignments 
    //static std::random_device seed;
    //static std::mt19937 random_number_generator(seed());
    /*
    std::uniform_int_distribution<size_t> indices(0, data.size() - 1);
    // Pick centroids as random points from the dataset.
    int index;
    for (int i=0; i<k; i++) {
      index = (int)(indices(random_number_generator)/CHANNEL_NUM);
      means.at(i) = data[index];
    }
    */

    Point* means_device;
    Point* points_device;
    size_t* assignments_device;
    uint8_t* new_img_device;
    uint8_t* rgb_img_device;
    uint8_t* new_img = (uint8_t*)malloc(sizeof(uint8_t) * height * width * CHANNEL_NUM);
    int* init_mean_nums;
    //printf("ENTERED");
    hipMalloc(&means_device, sizeof(Point) * k);
    hipMalloc(&points_device, sizeof(Point) * height * width );
    hipMalloc(&init_mean_nums, sizeof(int) * k );
    hipMalloc(&assignments_device, sizeof(size_t) * height * width);
    hipMalloc(&new_img_device, sizeof(uint8_t) * height * width*CHANNEL_NUM );
    hipMalloc(&rgb_img_device, sizeof(uint8_t) * height * width*CHANNEL_NUM );
    
    hipMemcpy(rgb_img_device, rgb_image, sizeof(uint8_t) * height * width*CHANNEL_NUM, hipMemcpyHostToDevice);
    //printf("COPIED");
    //hipMemcpy(&new_img_device, new_img, sizeof(uint8_t) * height * width, hipMemcpyDeviceToHost);
    /* Set seed */
    //CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL));

    /* Generate n floats on device */
    //hiprandGenerator_t gen;
    //CURAND_CALL(hiprandGenerateUniform(gen, init_mean_nums, height * width));
    //(Point* &points, Point* means, size_t* assignments, int number_of_iterations, int k, int height, int width, uint8_t* new_img, int* init_mean_nums)
    double start_time_exc = currentSeconds();
    k_means_main(kgridDim, kthreadsPerBlock, gridDim, threadsPerBlock, points_device, means_device, assignments_device, 
    number_of_iterations, k,  height, width, rgb_img_device, new_img_device, init_mean_nums);
    //printf("DONE");
    double end_time = currentSeconds();
    double duration_exc = end_time - start_time_exc;
    printf("Time: %f\n", duration_exc);
    hipMemcpy(new_img, new_img_device, sizeof(uint8_t) * height * width * CHANNEL_NUM, hipMemcpyDeviceToHost);

    //hipMemcpy(new_img, new_img_device, sizeof(Point) * height * width * CHANNEL_NUM, hipMemcpyDeviceToHost);
    stbi_write_png("cs_test1_out.png", width, height, CHANNEL_NUM, new_img, width*CHANNEL_NUM);  
    //printf("Finished k-means\n");
}

int main(int argc, char **argv){
    //printf("Starting off ... \n");
    const char *img_file = "cs_test1.jpg";
    int width, height, bpp;
    //printf("READING");
    uint8_t* rgb_image = stbi_load(img_file, &width, &height, &bpp, CHANNEL_NUM);  
    //printf("READ");
    //Point* df = get_df(rgb_image, width, height);
    
    k_means(rgb_image, width, height, 4, 2000);
    
    return 1;
    
}
